#include "memory_manager.cuh"
#include <hip/hip_runtime.h>

void printDeviceProperties() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, i));
        
        std::cout << "\n=== Device " << i << ": " << prop.name << " ===" << std::endl;
        std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "Total global memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
        std::cout << "Shared memory per block: " << prop.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "Constant memory: " << prop.totalConstMem / 1024 << " KB" << std::endl;
        std::cout << "Memory Clock Rate: " << prop.memoryClockRate / 1000 << " MHz" << std::endl;
        std::cout << "Memory Bus Width: " << prop.memoryBusWidth << " bits" << std::endl;
        std::cout << "L2 Cache Size: " << prop.l2CacheSize / 1024 << " KB" << std::endl;
        std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "Multiprocessor count: " << prop.multiProcessorCount << std::endl;
    }
}

float measureKernelTime(void (*kernel)(int), int size, int iterations) {
    // Warm-up
    kernel(size);
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Ölçüm
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    CUDA_CHECK(hipEventRecord(start));
    
    for (int i = 0; i < iterations; i++) {
        kernel(size);
    }
    
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    
    return milliseconds / iterations;
}