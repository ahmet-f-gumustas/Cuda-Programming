#include "hip/hip_runtime.h"
#include "memory_manager.cuh"

// Unified Memory kullanan vector addition
__global__ void vectorAddUnified(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < n; i += stride) {
        c[i] = a[i] + b[i];
    }
}

// Prefetch ile optimize edilmiş versiyon
void runUnifiedMemoryWithPrefetch(int size) {
    float *a, *b, *c;
    
    // Unified Memory ayır
    CUDA_CHECK(hipMallocManaged(&a, size * sizeof(float)));
    CUDA_CHECK(hipMallocManaged(&b, size * sizeof(float)));
    CUDA_CHECK(hipMallocManaged(&c, size * sizeof(float)));
    
    // CPU'da başlat
    for (int i = 0; i < size; i++) {
        a[i] = static_cast<float>(i);
        b[i] = static_cast<float>(i * 2);
    }
    
    // GPU device ID
    int device = 0;
    CUDA_CHECK(hipGetDevice(&device));
    
    // Prefetch to GPU
    CUDA_CHECK(hipMemPrefetchAsync(a, size * sizeof(float), device));
    CUDA_CHECK(hipMemPrefetchAsync(b, size * sizeof(float), device));
    CUDA_CHECK(hipMemPrefetchAsync(c, size * sizeof(float), device));
    
    // Kernel çalıştır
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    CUDA_CHECK(hipEventRecord(start));
    vectorAddUnified<<<gridSize, blockSize>>>(a, b, c, size);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float kernelTime;
    CUDA_CHECK(hipEventElapsedTime(&kernelTime, start, stop));
    
    std::cout << "    With prefetch: " << kernelTime << " ms" << std::endl;
    
    // Belleği temizle
    CUDA_CHECK(hipFree(a));
    CUDA_CHECK(hipFree(b));
    CUDA_CHECK(hipFree(c));
    
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

// Prefetch olmadan
void runUnifiedMemoryWithoutPrefetch(int size) {
    float *a, *b, *c;
    
    // Unified Memory ayır
    CUDA_CHECK(hipMallocManaged(&a, size * sizeof(float)));
    CUDA_CHECK(hipMallocManaged(&b, size * sizeof(float)));
    CUDA_CHECK(hipMallocManaged(&c, size * sizeof(float)));
    
    // CPU'da başlat
    for (int i = 0; i < size; i++) {
        a[i] = static_cast<float>(i);
        b[i] = static_cast<float>(i * 2);
    }
    
    // Kernel çalıştır (prefetch yok)
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    CUDA_CHECK(hipEventRecord(start));
    vectorAddUnified<<<gridSize, blockSize>>>(a, b, c, size);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float kernelTime;
    CUDA_CHECK(hipEventElapsedTime(&kernelTime, start, stop));
    
    std::cout << "    Without prefetch: " << kernelTime << " ms" << std::endl;
    
    // Belleği temizle
    CUDA_CHECK(hipFree(a));
    CUDA_CHECK(hipFree(b));
    CUDA_CHECK(hipFree(c));
    
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

void runUnifiedMemoryExample(int size) {
    std::cout << "  Unified Memory Performance:" << std::endl;
    
    // Prefetch olmadan
    runUnifiedMemoryWithoutPrefetch(size);
    
    // Prefetch ile
    runUnifiedMemoryWithPrefetch(size);
    
    // Memory advice örneği
    float *data;
    CUDA_CHECK(hipMallocManaged(&data, size * sizeof(float)));
    
    // Memory advice set et
    int device = 0;
    CUDA_CHECK(hipMemAdvise(data, size * sizeof(float), hipMemAdviseSetReadMostly, device));
    
    // Başlat
    for (int i = 0; i < size; i++) {
        data[i] = static_cast<float>(i);
    }
    
    // Basit bir kernel
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    
    auto sumKernel = [=] __device__ (int idx) {
        if (idx < size) {
            // Read-heavy operation
            float sum = 0.0f;
            for (int j = 0; j < 10; j++) {
                sum += data[idx];
            }
            data[idx] = sum / 10.0f;
        }
    };
    
    // Lambda kernel çalıştır
    auto kernel = [=] __global__ () {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        sumKernel(idx);
    };
    
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    CUDA_CHECK(hipEventRecord(start));
    kernel<<<gridSize, blockSize>>>();
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float adviceTime;
    CUDA_CHECK(hipEventElapsedTime(&adviceTime, start, stop));
    
    std::cout << "    With memory advice (ReadMostly): " << adviceTime << " ms" << std::endl;
    
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}